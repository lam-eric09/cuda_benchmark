
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <iostream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <cmath>

constexpr int N = 100000000;

__global__ void vector_add(double *out, double *a, double *b){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N) out[i] = a[i] + b[i];
}

int main(){
    int bytes = sizeof(double)*N;

    // Allocate pageable memory
    // a = new double[bytes];
    std::vector<double> a(N);
    std::vector<double> b(N);
    std::vector<double> out(N);

    for (int i=0; i<N; i++){
        a[i] = 1.0;
        b[i] = 2.0;
    }

    // Allocate device memory
    double *d_out, *d_a, *d_b;
    hipMalloc((void**)&d_out, bytes);
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);

    // Copy to device
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    // Run kernel
    int thread_per_block = 64;
    int total_blocks = ceil((N+thread_per_block-1)/thread_per_block);
    vector_add<<<total_blocks,thread_per_block>>>(d_out, d_a, d_b);

    hipMemcpy(out.data(), d_out, bytes, hipMemcpyDeviceToHost);

    std::cout << N-1 << out[N-1] << std::endl;

    hipFree(d_out);
    hipFree(d_a);
    hipFree(d_b);
    
    return 0;
}