
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <cmath>

#define N 100000000

__global__ void add_vector(double *out, double *a, double *b){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N) out[i] = a[i] + b[i];
}

int main(){
    double *out, *a, *b;
    int bytes = N*sizeof(double);

    // Allocate pinned non pageable memory in host
    hipHostMalloc((void**)&out, bytes, hipHostMallocDefault);
    hipHostMalloc((void**)&a, bytes, hipHostMallocDefault);
    hipHostMalloc((void**)&b, bytes, hipHostMallocDefault);

    // Allocate device memory
    double *d_out, *d_a, *d_b;
    hipMalloc((void**)&d_out, bytes);
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);
    
    // Initialize data
    for (int i=0; i<N; i++){
        a[i] = 1L;
        b[i] = 2L;
    }
        
    // Copy to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    // Run kernel 
    int thread_per_block = 64;
    int total_blocks = ceil((N+thread_per_block-1)/thread_per_block);
    add_vector<<<total_blocks,thread_per_block>>>(d_out, d_a, d_b);

    // Copy to host
    hipMemcpy(out, d_out, bytes, hipMemcpyDeviceToHost);

    std::cout << N-1 << out[N-1] << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(out);
    return 0;
}
